#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cuda_funcs.h"
#include "def.h"

#define BLOCK_SIZE  256

// #if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))

// extern char conservatives_arr[CONSERVATIVE_COUNT][CONSERVATIVE_MAX_LEN];
// extern char semi_conservatives_arr[SEMI_CONSERVATIVE_COUNT][SEMI_CONSERVATIVE_MAX_LEN];
extern char char_hash[NUM_CHARS][NUM_CHARS];

// #endif

__global__ void sumCommMultiBlock(double* scores, Mutant* mutants, int array_size, int is_max)
{
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*BLOCK_SIZE;
    const int gridSize = BLOCK_SIZE*gridDim.x;
    double s = is_max ? INT_MIN : INT_MAX;
    Mutant m;
    for (int i = gthIdx; i < array_size; i += gridSize)
    {   
        if ((is_max && scores[i] > s) || (!is_max && scores[i] < s))
        {
            s = scores[i];
            m = mutants[i];
        }
    }
    __shared__ double shScore[BLOCK_SIZE];
    __shared__ Mutant shMutant[BLOCK_SIZE];
    shScore[thIdx] = s;
    shMutant[thIdx] = m;
    __syncthreads();
    for (int size = BLOCK_SIZE/2; size>0; size/=2)
    { //uniform
        if (thIdx<size && thIdx + size < array_size)
        {
            if ((is_max && shScore[thIdx + size] >= shScore[thIdx]) ||
                (!is_max && shScore[thIdx + size] <= shScore[thIdx]))   //  include '==' to make sure the smaller offset is saved
            {
                if (shScore[thIdx + size] == shScore[thIdx] && !(shMutant[thIdx + size].offset < shMutant[thIdx].offset))   //  if scores equal and the smaller offset is save -> continue
                    continue;
                //  otherwise, the scores are not equal, or they are equal, but the greater one is saved -> save the new score
                shScore[thIdx] = shScore[thIdx + size];
                shMutant[thIdx] = shMutant[thIdx + size];
            }

        }
        __syncthreads();
    }
    if (thIdx == 0)
    {
        scores[blockIdx.x] = shScore[0];
        mutants[blockIdx.x] = shMutant[0];
    }
}

double gpu_run_program(ProgramData* data, Mutant* returned_mutant, int first_offset, int last_offset)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate memory on GPU to copy the data from the host
    ProgramData* gpu_data;
    Mutant* gpu_mutant;
    double* scores;
    double returned_score = -999;

    int offsets = last_offset - first_offset;

    err = hipMalloc(&gpu_data, sizeof(ProgramData));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(gpu_data, data, sizeof(ProgramData), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&gpu_mutant, offsets * sizeof(Mutant));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&scores, offsets * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 threadsPerBlockHash(NUM_CHARS, NUM_CHARS);
    dim3 numBlocksHash(1, 1);
    fill_hashtable_gpu<<<numBlocksHash, threadsPerBlockHash>>>();

    // Launch the Kernel
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (offsets + threadsPerBlock - 1) / threadsPerBlock;//offsets;
    printf("blocks=%d, threads=%d\n", blocksPerGrid, threadsPerBlock);
    get_best_mutant_gpu<<<blocksPerGrid, threadsPerBlock, 0>>>(gpu_data, gpu_mutant, scores, first_offset, last_offset);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    sumCommMultiBlock<<<blocksPerGrid, threadsPerBlock>>>(scores, gpu_mutant, last_offset - first_offset, data->is_max);
    sumCommMultiBlock<<<1, threadsPerBlock>>>(scores, gpu_mutant, blocksPerGrid, data->is_max);
    hipDeviceSynchronize();


    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // gpu_mutant[0] = gpu_mutant[10];
    // scores[0] = scores[10];

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    //  the best mutant is in index 0 in mutants array
    err = hipMemcpy(returned_mutant, &gpu_mutant[0], sizeof(Mutant), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy result mutant from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // it's score in index 0 in scores array
    err = hipMemcpy(&returned_score, &scores[0], sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy result score from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(gpu_data);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(gpu_mutant);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(scores);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("%2d, %2d, %f\n", returned_mutant->offset, returned_mutant->char_offset);


    return returned_score;
}


__global__ void get_best_mutant_gpu(ProgramData* data, Mutant* mutants, double* scores, int first_offset, int last_offset)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;        //  calculate thread index in the arrays
    // printf("%2d, %2d, %2d\n", blockDim.x, blockDim.y, blockDim.z);
    // printf("%2d, %2d, %2d\n", blockIdx.x, blockIdx.y, blockIdx.z);
    // printf("%2d, %2d, %2d\n", threadIdx.x, threadIdx.y, threadIdx.z);

    // printf("index=%3d val=%f\n", idx, scores[idx]);
    // printf("%c\n", char_hash_cuda[0][0]);
    // fill_hashtable_gpu();  


    int offsets = last_offset - first_offset;

    if (idx >= offsets)
    {
        scores[idx] = 0;
        return;
    }

    // mutants[idx]
    // scores[idx]

    scores[idx] = find_best_mutant_offset_gpu(data, first_offset + idx, &mutants[idx]);
    // printf("off=%2d charoff=%2d, %f, %f\n", mutants[idx].offset, mutants[idx].char_offset, scores[idx]);

    __syncthreads();
}

__device__ double find_best_mutant_offset_gpu(ProgramData* data, int offset, Mutant* mt)
{
    int idx1, idx2;
    double total_score = 0;
    double pair_score, mutant_diff, best_mutant_diff;
    int iterations = strlen_gpu(data->seq2);
    char c1, c2, subtitue;

    for (int i = 0; i < iterations; i++)            //  iterate over all the characters
    {
        idx1 = offset + i;                      //  index of seq1
        idx2 = i;                               //  index of seq2
        c1 = data->seq1[idx1];                  //  current char in seq1
        c2 = data->seq2[idx2];                  //  current char in seq2
        pair_score = get_weight(get_hash_sign(c1, c2), data->weights);    //  get weight before substitution
        total_score += pair_score;

        subtitue = find_char(c1, c2, data->weights, data->is_max);
        mutant_diff = get_weight(get_hash_sign(c1, subtitue), data->weights) - pair_score;    //  difference between original and mutation weights
        mutant_diff = abs(mutant_diff);


        if (mutant_diff > best_mutant_diff || i == 0)
        {
            best_mutant_diff = mutant_diff;
            mt->ch = subtitue;
            mt->char_offset = i;        //  offset of char inside seq2
            mt->offset = offset;
        }
    }
    if (data->is_max)
        return total_score + best_mutant_diff;
    return total_score - best_mutant_diff;     //  best mutant is returned in struct mt
}

__host__ __device__ char find_char(char c1, char c2, double* w, int is_max)
{
    char sign = get_hash_sign(c1, c2);

    return  is_max ?
            find_max_char(c1, c2, sign, w)   :
            find_min_char(c1, c2, sign, w);
}

__host__  __device__ char find_max_char(char c1, char c2, char sign, double* w)
{
    char ch;
    switch (sign)
    {
    case STAR:
        return c2;

    case DOT:                   //  if there is DOT between two characters, a START subtitution is possible
    case SPACE:                 //  if there is SPACE between two characters, a START subtitution is possible
        return c1;

    case COLON:
        double dot_diff = w[COLON_W] - w[DOT_W];
        double space_diff = w[COLON_W] - w[SPACE_W];

        if (!(dot_diff > 0 || space_diff > 0))      //  if both not greater than 0 (negative change or no change at all)
        {                                           //  then, no score change and return the same character
            return c2;
        }

        if (space_diff > dot_diff)                 //  if SPACE subtitution is better than DOT
        {
            ch = get_char_by_sign_with_restrictions(c1, SPACE, c2);
            if (ch != NOT_FOUND_CHAR)       //  if found SPACE subtitution
                return ch;
            
            //  if could not find SPACE subtitution, and DOT is better than no subtitution
            if (dot_diff > 0)
            {
                ch = get_char_by_sign_with_restrictions(c1, DOT, c2);
                if (ch != NOT_FOUND_CHAR)       //  if found DOT subtitution
                    return ch;
            }

            //  otherwise, no subtitution found
            return c2;
        }

        //  otherwise, it will try to find DOT subtitution
        ch = get_char_by_sign_with_restrictions(c1, DOT, c2);
        if (ch != NOT_FOUND_CHAR)       //  if found DOT subtitution
            return ch;

        //  if could not find DOT subtitution, and SPACE is better than no subtitution
        if (space_diff > 0)
        {
            ch = get_char_by_sign_with_restrictions(c1, SPACE, c2);
            if (ch != NOT_FOUND_CHAR)       //  if found SPACE subtitution
                return ch;
        }

        //  otherwise, no subtitution found
        return c2;
    }
    return c2;
}

__host__ __device__ char find_min_char(char c1, char c2, char sign, double* w)
{   
    char colon_sub = get_char_by_sign_with_restrictions(c1, COLON, c2);
    char dot_sub = get_char_by_sign_with_restrictions(c1, DOT, c2);
    char space_sub = get_char_by_sign_with_restrictions(c1, SPACE, c2);

    double colon_diff, dot_diff, space_diff;

    switch (sign)
    {
    case STAR:
        dot_diff = - w[STAR_W] - w[DOT_W];
        space_diff = - w[STAR_W] - w[SPACE_W];

        if (!(dot_diff < 0 || space_diff < 0))    //  if any subtitution will not decrease the score
            return c2;                                              //  than return the same letter and score

        if (dot_diff < space_diff)
        {
            if (dot_sub != NOT_FOUND_CHAR)
                return dot_sub;
        }

        //  could not find DOT subtitution
        if (space_diff < 0)
        {
            if (space_sub != NOT_FOUND_CHAR)
                return space_sub;

            //  could not find SPACE subtitution, but DOT might be better than nothing
            if (dot_diff < 0 && dot_sub != NOT_FOUND_CHAR)
                return dot_sub;
        }

        return c2;  //  could not find any subtitution
    
    case COLON:
        dot_diff = w[COLON_W] - w[DOT_W];
        space_diff = w[COLON_W] - w[SPACE_W];

        if (!(dot_diff < 0 || space_diff < 0))      //  if any subtitution will not decrease the score
            return c2;                              //  than return the same letter and score

        if (dot_diff < space_diff)                  //  if DOT subtitution is better than SPACE
        {
            if (dot_sub != NOT_FOUND_CHAR)          //  if found DOT subtitution
                return dot_sub;
        }

        if (space_diff < 0)
        {
            if (space_sub != NOT_FOUND_CHAR)
                return space_sub;

            //  could not find SPACE subtitution, but DOT might be better than nothing
            if (dot_diff < 0 && dot_sub != NOT_FOUND_CHAR)
                return dot_sub;
        }
        
        return c2;  // could not find any subtitution

    case DOT:
        colon_diff = w[DOT_W] - w[COLON_W];
        space_diff = w[DOT_W] - w[SPACE_W];

        if (!(colon_diff < 0 && space_diff < 0))    //  if any subtitution will not decrease the score
            return c2;                              //  than return the same letter and score

        if (colon_diff < space_diff)                //  if COLON subtitution is better than SPACE   
        {
            if (colon_sub != NOT_FOUND_CHAR)
                return colon_sub;
        }

        if (space_diff < 0)
        {
            if (space_sub != NOT_FOUND_CHAR)
                return space_sub;
            
            //  could not find SPACE subtitution, but COLON might still be better than nothing
            if (colon_diff < 0 && colon_sub != NOT_FOUND_CHAR)
                return colon_sub;
        }

        return c2;  // could not find any subtitution

    case SPACE:
        colon_diff = w[SPACE_W] - w[COLON_W];
        dot_diff = w[SPACE_W] - w[DOT_W];

        if (!(colon_diff < 0 && dot_diff < 0))      //  if any subtitution will not decrease the score
            return c2;                              //  than return the same letter and score

        if (colon_diff < dot_diff)                  //  if COLON subtitution is better than DOT
        {
            if (colon_sub != NOT_FOUND_CHAR)        //  if found COLON subtitution
                return colon_sub;
        }

        if (dot_diff < 0)
        {
            if (dot_sub != NOT_FOUND_CHAR)          //  if found DOT subtitution
                return dot_sub;

            //  could not find DOT subtitution, but COLON might still be better than nothing
            if (colon_diff < 0 && colon_sub != NOT_FOUND_CHAR)
                return colon_sub;
        }

        return c2;  // could not find any subtitution
    }
    return c2;      //  sign was not any of the legal signs
}

__host__ __device__ char get_char_by_sign_with_restrictions(char by, char sign, char rest)
{
    char last_char = FIRST_CHAR + NUM_CHARS;
    for (char ch = FIRST_CHAR; ch < last_char; ch++)   //  iterate over alphabet (A-Z)
    {
        if (get_hash_sign(by, ch) == sign && get_hash_sign(rest, ch) != COLON)  //  if found character which is not in the same conservative group with the previous one
            return ch;
    }
    return NOT_FOUND_CHAR;
}

__host__ __device__ char get_hash_sign(char c1, char c2)
{
    if (c1 > FIRST_CHAR + NUM_CHARS || c2 > FIRST_CHAR + NUM_CHARS)   return DASH;
    if (c1 == DASH && c2 == DASH)   return STAR;
    if (c1 == DASH || c2 == DASH)   return SPACE;

    if (c1 >= c2)       //  only the bottom triangle of the hash table is full -> (hash[x][y] = hash[y][x])
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
        return char_hash_cuda[c1 - FIRST_CHAR][c2 - FIRST_CHAR];
    return char_hash_cuda[c2 - FIRST_CHAR][c1 - FIRST_CHAR];
#else
        return char_hash[c1 - FIRST_CHAR][c2 - FIRST_CHAR];
    return char_hash[c2 - FIRST_CHAR][c1 - FIRST_CHAR];
#endif
}

__host__ __device__ double get_weight(char sign, double* w)
{
    switch (sign)
    {
    case STAR:  return w[STAR_W];
    case COLON: return -w[COLON_W];
    case DOT:   return -w[DOT_W];
    case SPACE: return -w[SPACE_W];
    }
    return 0;
}

__host__ __device__ int is_contain(char* str, char c)
{
    char* t = str;

    while (*t)
    {
        if (*t == c)
            return 1;
        
        ++t;
    }
    return 0;
}

//  check if both characters present in the same conservative group
__host__ __device__ int is_conservative(char c1, char c2)
{
    for (int i = 0; i < CONSERVATIVE_COUNT; i++)    //  iterate over the conservative groups
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
        if (is_contain(conservatives_arr_cuda[i], c1) && is_contain(conservatives_arr_cuda[i], c2))   //  if both characters present
#else
        if (is_contain(conservatives_arr[i], c1) && is_contain(conservatives_arr[i], c2))   //  if both characters present
#endif
            return 1;
    return 0;
}

//  check if both characters present in the same semi-conservative group
__host__ __device__ int is_semi_conservative(char c1, char c2)
{
    for (int i = 0; i < SEMI_CONSERVATIVE_COUNT; i++)   //  iterate over the semi-conservative groups
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
            if (is_contain(semi_conservatives_arr_cuda[i], c1) && is_contain(semi_conservatives_arr_cuda[i], c2))   //  if both characters present
#else
            if (is_contain(semi_conservatives_arr[i], c1) && is_contain(semi_conservatives_arr[i], c2))   //  if both characters present
#endif
                return 1;
    return 0;
}

__host__ __device__ char evaluate_chars(char a, char b)
{
    if      (a == b)                        return STAR;
    else if (is_conservative(a, b))         return COLON;
    else if (is_semi_conservative(a, b))    return DOT;

    return SPACE;
}

__device__ int strlen_gpu(char* str)
{
    int count = 0;
    char* t = str;
    while (*t)
    {
        ++count;
        ++t;
    }
    return count;
}

__global__ void fill_hashtable_gpu()
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    char c1 = FIRST_CHAR + row;
    char c2 = FIRST_CHAR + col;
    char_hash_cuda[row][col] = evaluate_chars(c1, c2);
}
