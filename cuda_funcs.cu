#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cuda_funcs.h"

__global__ void get_max_value_GPU(ProgramData* data, Mutant* my_mutant, int first_offset, int last_offset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    int num_elements = last_offset - first_offset;
    if (i >= num_elements)
        return;

    int my_offset = first_offset + i;
    // printf("gpu tid %4d, offset %3d\n", i, my_offset);
}


double gpu_run_program(ProgramData* data, Mutant* my_mutant, int first_offset, int last_offset)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate memory on GPU to copy the data from the host
    ProgramData* gpu_data;
    Mutant* gpu_mutant;

    err = hipMalloc((void **)&gpu_data, sizeof(ProgramData));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy data from host to the GPU memory
    err = hipMemcpy(gpu_data, data, sizeof(ProgramData), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&gpu_mutant, sizeof(Mutant));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int num_elements = last_offset - first_offset;

    // Launch the Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
    get_max_value_GPU<<<blocksPerGrid, threadsPerBlock, 0>>>(gpu_data, gpu_mutant, first_offset, last_offset);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory so the host could sum it up
    err = hipMemcpy(my_mutant, gpu_mutant, sizeof(Mutant), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(gpu_data) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(gpu_mutant) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}
